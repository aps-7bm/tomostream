#include "hip/hip_runtime.h"
#include "radonortho.cuh"
#include "kernels.cuh"
#include  <stdio.h>
radonortho::radonortho(size_t ntheta, size_t n, size_t nz, size_t nthetapi)
 : ntheta(ntheta), n(n), nz(nz), nthetapi(nthetapi) 
{
	// arrays allocation on GPU
	hipMalloc((void **)&fx, n * nz * sizeof(float));
	hipMalloc((void **)&fy, n * nz *sizeof(float));
	hipMalloc((void **)&fz, n * n *sizeof(float));
	hipMalloc((void **)&g, n * ntheta * nz * sizeof(float));
	hipMalloc((void **)&gs, n * ntheta * nz * sizeof(unsigned char));	
	hipMalloc((void **)&flat, n * nz * sizeof(float));
	hipMalloc((void **)&dark, n * nz * sizeof(float));
	
	hipMalloc((void **)&fg, (n / 2 + 1) * ntheta * nz * sizeof(float2));
	hipMalloc((void **)&filter, (n / 2 + 1) * sizeof(float));	
	hipMalloc((void **)&theta, ntheta * sizeof(float));

	hipMemset(fx, 0, n * nz * sizeof(float));
	hipMemset(fy, 0, n * nz * sizeof(float));
	hipMemset(fz, 0, n * n * sizeof(float));
	
	//fft plans for filtering
	int ffts[] = {n};
	int idist = n;
	int odist = n / 2 + 1;
	int inembed[] = {n};
	int onembed[] = {n / 2 + 1};
	hipfftPlanMany(&plan_forward, 1, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_R2C, ntheta * nz);
	hipfftPlanMany(&plan_inverse, 1, ffts, onembed, 1, odist, inembed, 1, idist, HIPFFT_C2R, ntheta * nz);
	
	//init thread blocks and block grids
	BS3d.x = 32;
	BS3d.y = 32;
	BS3d.z = 1;

	GS3d1.x = ceil(n / (float)BS3d.x);
	GS3d1.y = ceil(nz / (float)BS3d.y);
	GS3d1.z = ceil(ntheta / (float)BS3d.z);

	GS3d2.x = ceil(n / (float)BS3d.x);
	GS3d2.y = ceil(n / (float)BS3d.y);
	
	GS3d3.x = ceil(n / (float)BS3d.x);
	GS3d3.y = ceil(nz / (float)BS3d.y);

	is_free = false;	
}


// destructor, memory deallocation
radonortho::~radonortho() { free(); }


void radonortho::free()
{
	if (!is_free) 
	{
		hipFree(g);
		hipFree(gs);		
		hipFree(fg);
		hipFree(fx);
		hipFree(fy);
		hipFree(fz);
		hipFree(filter);

		hipFree(theta);
		hipfftDestroy(plan_forward);
		hipfftDestroy(plan_inverse);
		is_free = true;   
	}
	
}


void radonortho::rec(size_t fx_,size_t fy_,size_t fz_, size_t g_, size_t theta_, float center, int ix, int iy, int iz)
{
	for (int i=0;i<nthetapi/ntheta;i++)
	{
		// copy data and angles to GPU
		hipMemcpy(gs, &((unsigned char *)g_)[i*ntheta*nz*n], n * ntheta * nz * sizeof(unsigned char), hipMemcpyDefault);	
		hipMemcpy(theta, (float *)theta_, ntheta * sizeof(float), hipMemcpyDefault);
		
		// convert short to float, apply dark-flat field correction
		correction<<<GS3d1, BS3d>>>(g, gs, flat, dark, n, nz, ntheta);	

		// fft for filtering in the frequency domain
		hipfftExecR2C(plan_forward, (hipfftReal *)g, (hipfftComplex *)fg);
		// fbp filtering
		applyfilter<<<GS3d1, BS3d>>>(fg, filter, n, nz, ntheta);
		// fft back
		hipfftExecC2R(plan_inverse, (hipfftComplex *)fg, (hipfftReal *)g);
		
		// reconstruct slices via summation over lines	
		orthox<<<GS3d3, BS3d>>>(fx, g, theta, center, ix, n, nz, ntheta);
		orthoy<<<GS3d3, BS3d>>>(fy, g, theta, center, iy, n, nz, ntheta);
		orthoz<<<GS3d2, BS3d>>>(fz, g, theta, center, iz, n, nz, ntheta);
	}			
		
	//copy result to cpu
	hipMemcpy((float *)fx_, fx, n * nz * sizeof(float), hipMemcpyDefault);
	hipMemcpy((float *)fy_, fy, n * nz * sizeof(float), hipMemcpyDefault);
	hipMemcpy((float *)fz_, fz, n * n * sizeof(float), hipMemcpyDefault);	
}

void radonortho::set_filter(size_t filter_)
{
	hipMemcpy(filter, (float*) filter_, (n/2+1)*sizeof(float),hipMemcpyDefault);
}

void radonortho::set_flat(size_t flat_)
{
	hipMemcpy(flat, (float*) flat_, n*nz*sizeof(float),hipMemcpyDefault);
	
}

void radonortho::set_dark(size_t dark_)
{
	hipMemcpy(dark, (float*) dark_, n*nz*sizeof(float),hipMemcpyDefault);
	
}